#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void add_kernel(float* a, float* b, float* c, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

void add_cuda(torch::Tensor a, torch::Tensor b, torch::Tensor c) {
    int size = a.size(0);
    int threads = 1024;
    int blocks = (size + threads - 1) / threads;

    add_kernel<<<blocks, threads>>>(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(), size);
}